#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <time.h>

using namespace std;

#define cudaCheckError() {																\
	hipError_t e=hipGetLastError();													\
	if(e!=hipSuccess) {																\
		printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));	\
		}																					\
}


inline int GetBlockSize(int b, int maxSize)
{
	if (b <= maxSize)
		return b;
	else
		return maxSize;
}


inline int GetGridSize(int n, int b)
{
	if (n%b == 0)
		return n / b;
	else
		return int(n*1.0 / (b*1.0)) + 1;
}

__device__ float EvalMaxGPU(float a, float b) {
	return ((a) > (b)) ? (a) : (b);
}

__global__ void FindMax4(float *in, float *out, int n)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if (i < n && j < n)
	{
		// Output index
		int index = j * n + i;
	

		// Input index
		// Set up the input index correctly for the 4 inputs
		int index1 = i*j;
		int index2 = i*j+n/2;
		int index3 = i*j+n*n/2;
		int index4 = i*j+n*n/2 + n/2;
		
		//int index1 = i*j;
		//int index2 = i*j +1;
		//int index3 = i*j + n;
		//int index4 = i*j + n+1;
		// Compute the max of 4 values
		float max1 = EvalMaxGPU(in[index1], in[index2]);
		float max2 = EvalMaxGPU(in[index3], in[index4]);
		float max = EvalMaxGPU(max1, max2);
		out[index] = max;
	}
}

void InitMatrix(float* a, int n, int m)
{
	srand((int)time(NULL));
	for (int j = 0; j < m; j++)
		for (int i = 0; i < n; i++)
			a[j*n + i] = float(10.0 * rand() / (RAND_MAX*1.0));
	
	// Use the code below for debugging if required
		//a[j*n + i] = j*n + i;

}

void PrintMatrix(float* a, int n, int m)
{
	for (int j = 0; j < m; j++)
	{
		for (int i = 0; i < n; i++)
			cout << a[j*n + i] << " ";
		cout << endl;
	}
}
float FindMaxCPU(float* a, int n, int m)
{ 
	float maxVal = 0;
	for (int j = 0; j < m; j++)
		for (int i = 0; i < n; i++)
			maxVal = max(maxVal, a[j*n + i]);
	return maxVal;
}

int main()
{
	clock_t t1;
	clock_t t;

	// Set size of the matrix
	int n = 10;

	// Create CPU Array
	float* matrix = new float[n*n];
	InitMatrix(matrix, n, n);
	cout << "Created a " << n << " x " << n << " Matrix." << endl;

	float maxVal = FindMaxCPU(matrix, n, n);
	cout << "Maximum value from CPU computation is : " << maxVal << endl;
	t1 = clock() - t1;
	printf("It took me %d clicks (%f seconds).\n",t1,((float)t1)/CLOCKS_PER_SEC);

	// Use the following code for print debugging
	//cout << endl;
	//PrintMatrix(matrix, n, n);
	//cout << endl;

	// Allocate GPU Memory
	float* matrix1CUDA;
	float* matrix2CUDA;
	hipMalloc((void**)&(matrix1CUDA), n*n*sizeof(float));
	hipMalloc((void**)&(matrix2CUDA), n*n*sizeof(float));
	cudaCheckError();

	// Copy GPU Memory
	hipMemcpy(matrix1CUDA, matrix, n*n*sizeof(float), hipMemcpyHostToDevice);
	cudaCheckError();

	// Setup swap of CUDA device pointers
	float* inputCUDA;
	float* outputCUDA;
	inputCUDA = (matrix1CUDA);
	outputCUDA = (matrix2CUDA);

	// Run the Kernel
	for (int p = n / 2; p >= 1; p = p / 2)
	{
		dim3 block(GetBlockSize(p, 2), GetBlockSize(p, 1), 1);
		dim3 grid(GetGridSize(p, block.x), GetGridSize(p, block.y), 1);
		FindMax4 << < grid, block >> >(inputCUDA, outputCUDA, p);
		cudaCheckError();
	
	
		cout<<p<<endl;
		
		// Use the following code for print debugging
#ifdef DEBUG
		//float* tempDataIn = new float[2 * p * 2 * p];
		//hipMemcpy(tempDataIn, inputCUDA, 2 * p * 2 * p * sizeof(float), hipMemcpyDeviceToHost);
		//cudaCheckError();
		//PrintMatrix(tempDataIn, 2*p, 2*p);
		//cout << endl;
		//delete[] tempDataIn;

		//float* tempDataOut = new float[p*p];
		//hipMemcpy(tempDataOut, outputCUDA, p * p * sizeof(float), hipMemcpyDeviceToHost);
		//cudaCheckError();
		//PrintMatrix(tempDataOut, p, p);
		//cout << endl;
		//delete[] tempDataOut;
#endif

		// Swap input output pointers
		float* oldInputCUDA = inputCUDA;
		inputCUDA = outputCUDA;
		outputCUDA = oldInputCUDA;

	}

	float maxValGPU;
	hipMemcpy(&maxValGPU, inputCUDA, 1 * sizeof(float), hipMemcpyDeviceToHost);
	cout << "Maximum value from GPU computation is : " << maxValGPU << endl;

	hipDeviceSynchronize();
	cudaCheckError();

	// Free the Memory
	hipFree(matrix1CUDA);
	hipFree(matrix2CUDA);
#ifdef DEBUG
	cudaCheckError();
#endif
	t = clock() - t;
	printf("It took me %d clicks (%f seconds).\n",t,((float)t)/CLOCKS_PER_SEC);
	return 0;
}
